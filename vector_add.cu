#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

int main() {
  int n = 1000000;
  size_t size = n * sizeof(int);
  int *h_a, *h_b, *h_c;
  int *d_a, *d_b, *d_c;

  // Allocate host memory
  h_a = (int *)malloc(size);
  h_b = (int *)malloc(size);
  h_c = (int *)malloc(size);

  // Initialize host arrays
  for (int i = 0; i < n; i++) {
    h_a[i] = i;
    h_b[i] = i * 2;
  }

  // Allocate device memory
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Copy input data from host to device memory
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // Launch kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

  // Copy result from device to host
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // Verify result
  for (int i = 0; i < n; i++) {
    if (h_c[i] != h_a[i] + h_b[i]) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(1);
    }
  }

  printf("Test PASSED\n");

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Free host memory
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}